/**
    Author: Dimitriadis Vasileios 8404
    Faculty of Electrical and Computer Engineering AUTH
    3rd assignment at Parallel and Distributed Systems (7th semester)
    This is a parallel implementation of mean shift algorithm using the
    Gaussian probability density function.
  **/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define N 60000
#define DIMENSIONS 5
#define EPSILON 0.001
#define VAR 0.001 // =σ^2 variance
#define N_Threads 1024

struct timeval startwtime, endwtime;
double seq_time;


void getinput(double *x, char *filename);
__global__ void meanshift(double *dev_x, double *dev_y, int dim, double eps, double var);
__device__ double find_distance(double *y, int i, double *x, int j, int dim);
void show_results(double *y_new);

int main(int argc, char **argv)
{
  if (argc != 2)
  {
    printf("Need as input a dataset to process\n");
    exit (1);
  }

  double *x = (double *)malloc(N * DIMENSIONS * sizeof(double));
  if (x == NULL)
  {
    printf("Failed to allocate data at x...\n");
    exit(1);
  }
  getinput(x, argv[1]);

  double *y = (double *)malloc(N * DIMENSIONS * sizeof(double));
  if (y == NULL)
  {
    printf("Failed to allocate data at y...\n");
    exit(1);
  }

  double *dev_x;
  hipMalloc(&dev_x, N * DIMENSIONS * sizeof(double));

  double *dev_y;
  hipMalloc(&dev_y, N * DIMENSIONS * sizeof(double));

  hipMemcpy(dev_x, x, N * DIMENSIONS * sizeof(double), hipMemcpyHostToDevice);

  //Initialize y as x in gpu.
  hipMemcpy(dev_y, x, N * DIMENSIONS * sizeof(double), hipMemcpyHostToDevice);

  hipError_t error;
  size_t shared_size = N_Threads * DIMENSIONS + N_Threads;
  gettimeofday (&startwtime, NULL);
  meanshift<<<N, N_Threads, sizeof(double) * shared_size>>>(dev_x, dev_y, DIMENSIONS, EPSILON, VAR);
  gettimeofday (&endwtime, NULL);
  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
                        + endwtime.tv_sec - startwtime.tv_sec);

  hipMemcpy(y, dev_y, N * DIMENSIONS * sizeof(double), hipMemcpyDeviceToHost);
  error = hipGetLastError();
  if (error != hipSuccess)
  {
    printf("Error at copying back: %s\n", hipGetErrorString(error));
    exit(1);
  }
  hipDeviceSynchronize();
  error = hipGetLastError();
  if (error != hipSuccess)
  {
    printf("Error at Sync: %s\n", hipGetErrorString(error));
    exit(1);
  }
  printf("Time needed for mean shift is %f sec\n", seq_time);
  show_results(y);
  free(x);
  free(y);
  hipFree(dev_x);
  hipFree(dev_y);

  return (0);
}

void getinput(double *x, char *filename)
{
  FILE *fin;
  int i = 0, j;
  char *str = (char *)malloc(2 * DIMENSIONS * sizeof(double));
  char *token = (char *)malloc(sizeof(double));
  fin = fopen(filename, "r");
  if (fin == NULL)
  {
    printf("Error opening the file...");
    exit(1);
  }
  str = fgets(str, 2 * DIMENSIONS * sizeof(double), fin); //Take one point.
  while (str != NULL && i < N)
  {
    token = strtok(str, "\t"); //get one dimension per recursion.
    j = 0;
    while (token != NULL && j < DIMENSIONS)
    {
      x[i*DIMENSIONS + j] = atof(token);
      token = strtok(NULL, "\t");
      j++;
    }
    str = fgets(str, 2 * DIMENSIONS * sizeof(double), fin);
    i++;
  }
  fclose(fin);
  free(str);
  free(token);
}


__global__
void meanshift(double *dev_x, double *dev_y, int dim, double eps, double var)
{
  int start, end;
  // Every block is finding the new y until convergence.
  int i = blockIdx.x;
  int j = threadIdx.x;
  int n = gridDim.x;
  int n_th = blockDim.x;
  /** Every thread is processing a chunk of the data in order
      to find distances between y_i and all x faster. If the
      number of elements is devided equally by the number of
      threads then the chunk is N/(# of Blocks). If it is not then
      the first N%(# of Blocks) have one more element to process.
  **/
  int chunk = n  / n_th;
  if ((n % n_th) != 0)
  {
    if (j < (n % n_th))
    {
      chunk = chunk + 1;
      start = chunk * j;
      end = start + chunk;
    }
    else
    {
      start = chunk * j + (n % n_th);
      end = start + chunk;
    }
  }
  else
  {
    start = chunk * j;
    end = start + chunk;
  }
  /** Each block has its own shared memory and the
      size of it is number of threads multiplied by
      (dimensions + 1) to store the values of nominators
       and denominator that each thread finds.
  **/
  extern __shared__ double s[];
  double *nominator = &s[0];
  double *denominator = &s[n_th * dim];
  __shared__ int converge;
  converge = 0;
  double distance = 0, k;
  int l, r;
  while (!converge)
  {
    //Initialize nominators and denominators as 0.
    for (r=0; r<dim; r++)
    {
      nominator[j*dim + r] = 0;
    }
    denominator[j] = 0;
    // Every thread is responsible of finding the new nominators
    // and denominator in it's chunk.
    for (l=start; l<end; l++)
    {
      distance = find_distance(dev_y, i, dev_x, l, dim);
      if (sqrt(distance) <= var)
      {
        k = exp(-distance / (2 * var)); //Guassian possibility density function.
      }
      else
      {
        k = 0;
      }
      for (r=0; r<dim; r++)
      {
        nominator[j*dim + r] += k * dev_x[l*dim + r];
      }
      denominator[j] += k;
    }
    __syncthreads();
    // Reduction
    for (l=n_th/2; l>0; l>>=1)
    {
      if (j < l)
      {
        for (r=0; r<dim; r++)
        {
          nominator[j*dim + r] += nominator[(j+l) * dim + r];
        }
        denominator[j] += denominator[j+l];
      }
      __syncthreads();
    }
    // Threads from 0 to dim-1 store in the first column
    // of nominator the values of new y
    if (j < dim)
    {
      nominator[j] = nominator[j] / denominator[0];
    }
    __syncthreads();
    // Only first thread checking the converge.
    if (j == 0)
    {
      distance = 0;
      for (r=0; r<dim; r++)
      {
        distance += pow(dev_y[i*dim + r] - nominator[r], 2);
      }
      if (sqrt(distance) < eps)
      {
        converge = 1;
      }
    }
    __syncthreads();
    // New y is stored in place of the previous y.
    if (j < dim)
    {
      dev_y[i*dim + j] = nominator[j];
    }
    __syncthreads();
  }
}

__device__
double find_distance(double *y, int i, double *x, int j, int dim)
{
  double distance = 0;
  for (int l=0; l<dim; l++)
  {
    distance = distance + pow(y[i*dim + l]-x[j*dim + l], 2);
  }
  return distance;
}

void show_results(double *y_new)
{
  int i,j;
  for(i=0; i<20; i++)
  {
    for (j=0; j<DIMENSIONS; j++)
    {
      printf("%f ", y_new[i*DIMENSIONS + j]);
    }
    printf("\n");
  }
}
